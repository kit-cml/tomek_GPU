#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// #include "modules/drug_sim.hpp"
#include "modules/glob_funct.hpp"
#include "modules/glob_type.hpp"
#include "modules/gpu.cuh"
#include "modules/cipa_t.cuh"

#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <ctime>
#include <iostream>
#include <math.h>
#include <vector>
#include <sys/stat.h>

#define ENOUGH ((CHAR_BIT * sizeof(int) - 1) / 3 + 2)
char buffer[255];


// unsigned int datapoint_size = 7000;
const unsigned int sample_limit = 10000;


clock_t START_TIMER;

clock_t tic();
void toc(clock_t start = START_TIMER);

clock_t tic()
{
    return START_TIMER = clock();
}

void toc(clock_t start)
{
    std::cout
        << "Elapsed time: "
        << (clock() - start) / (double)CLOCKS_PER_SEC << "s"
        << std::endl;
}
  
int gpu_check(unsigned int datasize){
    int num_gpus;
    float percent;
    int id;
    size_t free, total;
    hipGetDeviceCount( &num_gpus );
    for ( int gpu_id = 0; gpu_id < num_gpus; gpu_id++ ) {
        hipSetDevice( gpu_id );
        hipGetDevice( &id );
        hipMemGetInfo( &free, &total );
        percent = (free/(float)total);
        printf("GPU No %d\nFree Memory: %ld, Total Memory: %ld (%f percent free)\n", id,free,total,percent*100.0);
    }
    percent = 1.0-(datasize/(float)total);
    //// this code strangely gave out too small value, so i disable the safety switch for now

    // printf("The program uses GPU No %d and %f percent of its memory\n", id,percent*100.0);
    // printf("\n");
    // if (datasize<=free) {
    //   return 0;
    // }
    // else {
    //   return 1;
    // }


    return 0;
    
}


// get the IC50 data from file
drug_t get_IC50_data_from_file(const char* file_name);
// return error and message based on the IC50 data
int check_IC50_content(const drug_t* ic50, const param_t* p_param);


int get_IC50_data_from_file(const char* file_name, double *ic50)
  {
  FILE *fp_drugs;
  char *token;
  char buffer_ic50[255];
  unsigned int idx;

  if( (fp_drugs = fopen(file_name, "r")) == NULL){
    printf("Cannot open file %s\n",
      file_name);
    return 0;
  }
  idx = 0;
  int sample_size = 0;
  fgets(buffer_ic50, sizeof(buffer_ic50), fp_drugs); // skip header
  while( fgets(buffer_ic50, sizeof(buffer_ic50), fp_drugs) != NULL )
  { // begin line reading
    token = strtok( buffer_ic50, "," );
    while( token != NULL )
    { // begin data tokenizing
      ic50[idx++] = strtod(token, NULL);
      token = strtok(NULL, ",");
    } // end data tokenizing
    sample_size++;
  } // end line reading

  fclose(fp_drugs);
  return sample_size;
  }


int get_cvar_data_from_file(const char* file_name, unsigned int limit, double *cvar)
{
  // buffer for writing in snprintf() function
  char buffer_cvar[255];
  FILE *fp_cvar;
  // cvar_t cvar;
  char *token;
  // std::array<double,18> temp_array;
  unsigned int idx;

  if( (fp_cvar = fopen(file_name, "r")) == NULL){
    printf("Cannot open file %s\n",
      file_name);
  }
  idx = 0;
  int sample_size = 0;
  fgets(buffer_cvar, sizeof(buffer_cvar), fp_cvar); // skip header
  while( (fgets(buffer_cvar, sizeof(buffer_cvar), fp_cvar) != NULL) && (sample_size<limit))
  { // begin line reading
    token = strtok( buffer_cvar, "," );
    while( token != NULL )
    { // begin data tokenizing
      cvar[idx++] = strtod(token, NULL);
      token = strtok(NULL, ",");
    } // end data tokenizing
    // printf("\n");
    sample_size++;
    // cvar.push_back(temp_array);
  } // end line reading

  fclose(fp_cvar);
  return sample_size;
}

int get_init_data_from_file(const char* file_name, double *init_states)
{
  // buffer for writing in snprintf() function
  char buffer_cache[1023];
  FILE *fp_cache;
  // cvar_t cvar;
  char *token;
  // std::array<double,18> temp_array;
  unsigned long idx;

  if( (fp_cache = fopen(file_name, "r")) == NULL){
    printf("Cannot open file %s\n",
      file_name);
  }
  idx = 0;
  unsigned int sample_size = 0;
  // fgets(buffer_cvar, sizeof(buffer_cvar), fp_cvar); // skip header
  while( (fgets(buffer_cache, sizeof(buffer_cache), fp_cache) != NULL) )
  { // begin line reading
    token = strtok( buffer_cache, "," );
    while( token != NULL )
    { // begin data tokenizing
      init_states[idx++] = strtod(token, NULL);
      // if(idx < 82){
      //     printf("%d: %lf\n",idx-1,init_states[idx-1]);
      // }
      token = strtok(NULL, ",");
    } // end data tokenizing
    // printf("\n");
    sample_size++;
    // cvar.push_back(temp_array);
  } // end line reading

  fclose(fp_cache);
  return sample_size;
}
int exists(const char *fname)
{
    FILE *file;
    if ((file = fopen(fname, "r")))
    {
        fclose(file);
        return 1;
    }
    // fclose(file);
    return 0;
}

int check_IC50_content(const drug_t* ic50, const param_t* p_param)
{
	if(ic50->size() == 0){
		printf("Something problem with the IC50 file!\n");
		return 1;
	}
	else if(ic50->size() > 2000){
		printf( "Too much input! Maximum sample data is 2000!\n");
		return 2;
	}
	else if(p_param->pace_max < 750 && p_param->pace_max > 1000){
		printf("Make sure the maximum pace is around 750 to 1000!\n");
		return 3;
	}
	// else if(mympi::size > ic50->size()){
	// 	printf("%s\n%s\n",
  //               "Overflow of MPI Process!",
  //               "Make sure MPI Size is less than or equal the number of sample");
	// 	return 4;
	// }
	else{
		return 0;
	}
}

int main(int argc, char **argv)
 {

    // for qinwards calculation
    double inal_auc_control = -90.547322;    // AUC of INaL under control model
    double ical_auc_control = -105.935067;   // AUC of ICaL under control model

    // input variables for cell simulation
    param_t *p_param = new param_t();  // input data for CPU
    param_t *d_p_param;  // input data for GPU parsing
	
  	p_param->init();
    edison_assign_params(argc,argv,p_param);
    p_param->show_val();

    double* ic50 = (double *)malloc(14 * sample_limit * sizeof(double));
    
    // if (p_param->is_cvar == true) cvar = (double *)malloc(18 * sample_limit * sizeof(double));
    double* cvar = (double *)malloc(18 * sample_limit * sizeof(double));  // conductance variability

    const int num_of_constants = 163;
    const int num_of_states = 43;
    const int num_of_algebraic = 223;
    const int num_of_rates = 43;
    const double CONC = p_param->conc;

    ////////// if we are in write time series mode (post processing) //////////
    if(p_param->is_time_series == 1 /*&& exists(p_param->cache_file) == 1 <- still unstable*/){

      printf("Using cached initial state from previous result!!!! \n\n");

      const unsigned int datapoint_size = p_param->sampling_limit; // sampling_limit: limit of num of data points in one sample
    
      double* cache = (double *)malloc((num_of_states+2) * sample_limit * sizeof(double)); // array for in silico results
      

      static const int CALCIUM_SCALING = 1000000;
      static const int CURRENT_SCALING = 1000;

      // snprintf(buffer, sizeof(buffer),
      //   "./drugs/bepridil/IC50_samples.csv"
      //   // "./drugs/bepridil/IC50_optimal.csv"
      //   // "./IC50_samples.csv"
      //   );

      int sample_size = get_IC50_data_from_file(p_param->hill_file, ic50);
      if(sample_size == 0)
          printf("Something problem with the IC50 file!\n");
      // else if(sample_size > 2000)
      //     printf("Too much input! Maximum sample data is 2000!\n");
      printf("Sample size: %d\n",sample_size);
      printf("Set GPU Number: %d\n",p_param->gpu_index);

      hipSetDevice(p_param->gpu_index);  // select a specific GPU

      if(p_param->is_cvar == true){
        int cvar_sample = get_cvar_data_from_file(p_param->cvar_file,sample_size,cvar);
        printf("Reading: %d Conductance Variability samples\n",cvar_sample);
      }

      printf("preparing GPU memory space \n");

        // char buffer_cvar[255];
        // snprintf(buffer_cvar, sizeof(buffer_cvar),
        // "./result/66_00.csv"
        // // "./drugs/optimized_pop_10k.csv"
        // );
        int cache_num = get_init_data_from_file(p_param->cache_file,cache);  //

        printf("Found cache for %d samples\n",cache_num);
        // note to self:
        // num of states+2 gave you at the very end of the file (pace number)
        // the very beginning -> the core number
      //   for (int z = 0; z <  num_of_states; z++) {printf("%lf\n", cache[z+1]);}
      //   printf("\n");
      //   for (int z = 0; z <  num_of_states; z++) {printf("%lf\n", cache[ 1*(num_of_states+2) + (z+2)]);}
      //   printf("\n");
      //   for (int z = 0; z <  num_of_states; z++) {printf("%lf\n", cache[ 2*(num_of_states+2) + (z+3)]);}
      // return 0 ;
      double *d_ic50;
      double *d_cvar;
      double *d_ALGEBRAIC;
      double *d_CONSTANTS;
      double *d_RATES;
      double *d_STATES;
      double *d_STATES_cache;
      // actually not used but for now, this is only for satisfiying the GPU regulator parameters
      double *d_STATES_RESULT;
      double *d_all_states;
      hipMalloc(&d_ALGEBRAIC, num_of_algebraic * sample_size * sizeof(double));
      hipMalloc(&d_CONSTANTS, num_of_constants * sample_size * sizeof(double));
      hipMalloc(&d_RATES, num_of_rates * sample_size * sizeof(double));
      hipMalloc(&d_STATES, num_of_states * sample_size * sizeof(double));
      hipMalloc(&d_STATES_cache, (num_of_states+2) * sample_size * sizeof(double));
      hipMalloc(&d_p_param,  sizeof(param_t));

      double *time;
      double *dt;
      double *states;
      double *ical;
      double *inal;
      double *cai_result;
      double *ina;
      double *ito;
      double *ikr;
      double *iks;
      double *ik1;
      cipa_t *temp_result, *cipa_result;
      // prep for 1 cycle plus a bit (7000 * sample_size)
      hipMalloc(&temp_result, sample_size * sizeof(cipa_t));  // for temporal ??
      hipMalloc(&cipa_result, sample_size * sizeof(cipa_t));  // output of postprocessing

      hipMalloc(&time, sample_size * datapoint_size * sizeof(double)); 
      hipMalloc(&dt, sample_size * datapoint_size * sizeof(double)); 

      hipMalloc(&states, sample_size * datapoint_size * sizeof(double));
      hipMalloc(&ical, sample_size * datapoint_size * sizeof(double));
      hipMalloc(&inal, sample_size * datapoint_size * sizeof(double));
      hipMalloc(&cai_result, sample_size * datapoint_size * sizeof(double));
      hipMalloc(&ina, sample_size * datapoint_size * sizeof(double));
      hipMalloc(&ito, sample_size * datapoint_size * sizeof(double));
      hipMalloc(&ikr, sample_size * datapoint_size * sizeof(double));
      hipMalloc(&iks, sample_size * datapoint_size * sizeof(double));
      hipMalloc(&ik1, sample_size * datapoint_size * sizeof(double));
      // hipMalloc(&d_STATES_RESULT, (num_of_states+1) * sample_size * sizeof(double));
      // hipMalloc(&d_all_states, num_of_states * sample_size * p_param->find_steepest_start * sizeof(double));
      hipMalloc(&d_ic50, sample_size * 14 * sizeof(double));  // ic50s of 7 channels 
      hipMalloc(&d_cvar, sample_size * 18 * sizeof(double));  // conductances of 18

      printf("Copying sample files to GPU memory space \n");
      hipMemcpy(d_STATES_cache, cache, (num_of_states+2) * sample_size * sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(d_ic50, ic50, sample_size * 14 * sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(d_cvar, cvar, sample_size * 18 * sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(d_p_param, p_param, sizeof(param_t), hipMemcpyHostToDevice);

      // // Get the maximum number of active blocks per multiprocessor
      // hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, do_drug_sim_analytical, threadsPerBlock);

      // // Calculate the total number of blocks
      // int numTotalBlocks = numBlocks * cudaDeviceGetMultiprocessorCount();
      tic();
      printf("Timer started, doing simulation.... \n\n\nGPU Usage at this moment: \n");
      int thread;
      if (sample_size>=32) thread = 32;// optimal number of thread by experience -> might be different for each GPU, can be 16, can be 32
      else thread = sample_size;
      // int block = int(ceil(sample_size*1.0/thread)+1);
      int block = (sample_size + thread - 1) / thread;
      // int block = (sample_size + thread - 1) / thread;
      if(gpu_check(15 * sample_size * sizeof(double) + sizeof(param_t)) == 1){
        printf("GPU memory insufficient!\n");
        return 0;
      }
      printf("Sample size: %d\n",sample_size);
      hipSetDevice(p_param->gpu_index);
      printf("\n   Configuration: \n\n\tblock\t||\tthread\n---------------------------------------\n  \t%d\t||\t%d\n\n\n", block,thread);
      // initscr();
      // printf("[____________________________________________________________________________________________________]  0.00 %% \n");


      kernel_DrugSimulation<<<block,thread>>>(d_ic50, d_cvar, d_CONSTANTS, d_STATES, d_STATES_cache, d_RATES, d_ALGEBRAIC, 
                                                d_STATES_RESULT, d_all_states,
                                                time, states, dt, cai_result,
                                                ina, inal, 
                                                ical, ito,
                                                ikr, iks, 
                                                ik1,
                                                sample_size,
                                                temp_result, cipa_result,
                                                d_p_param
                                                );
                                        //block per grid, threads per block
      // endwin();
      
      hipDeviceSynchronize();
      

      printf("allocating memory for computation result in the CPU, malloc style \n");
      double *h_states,*h_time,*h_dt,*h_ical,*h_inal,*h_cai_result,*h_ina,*h_ito,*h_ikr,*h_iks,*h_ik1;
      cipa_t *h_cipa_result;

      h_states = (double *)malloc(datapoint_size * sample_size * sizeof(double));
      printf("...allocated for STATES, \n");
      h_time = (double *)malloc(datapoint_size * sample_size * sizeof(double));
      printf("...allocated for time, \n");
      h_dt = (double *)malloc(datapoint_size * sample_size * sizeof(double));
      printf("...allocated for dt, \n");
      h_cai_result= (double *)malloc(datapoint_size * sample_size * sizeof(double));
      printf("...allocated for Cai, \n");
      h_ina= (double *)malloc(datapoint_size * sample_size * sizeof(double));
      printf("...allocated for iNa, \n");
      h_ito= (double *)malloc(datapoint_size * sample_size * sizeof(double));
      printf("...allocated for ito, \n");
      h_ikr= (double *)malloc(datapoint_size * sample_size * sizeof(double));
      printf("...allocated for ikr, \n");
      h_iks= (double *)malloc(datapoint_size * sample_size * sizeof(double));
      printf("...allocated for iks, \n");
      h_ik1= (double *)malloc(datapoint_size * sample_size * sizeof(double));
      printf("...allocated for ik1, \n");
      h_ical= (double *)malloc(datapoint_size * sample_size * sizeof(double));
      printf("...allocated for ICaL, \n");
      h_inal = (double *)malloc(datapoint_size * sample_size * sizeof(double));

      h_cipa_result = (cipa_t *)malloc( sample_size * sizeof(cipa_t));
      printf("...allocating for INaL and postprocessing, all set!\n");

      ////// copy the data back to CPU, and write them into file ////////
      printf("copying the data back to the CPU \n");

      hipMemcpy(h_states, states, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(h_time, time, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(h_dt, dt, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(h_ical, ical, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(h_inal, inal, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(h_cai_result, cai_result, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(h_ina, ina, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(h_ito, ito, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(h_ikr, ikr, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(h_iks, iks, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(h_ik1, ik1, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
      
      hipMemcpy(h_cipa_result, cipa_result, sample_size * sizeof(cipa_t), hipMemcpyDeviceToHost);
      
      FILE *writer;
      int check;
      bool folder_created = false;

      printf("writing to file... \n");
      // sample loop
      for (int sample_id = 0; sample_id<sample_size; sample_id++){
        // printf("writing sample %d... \n",sample_id);
        char sample_str[ENOUGH];
        char conc_str[ENOUGH];
        char filename[500] = "./result/";
        sprintf(sample_str, "%d", sample_id);
        sprintf(conc_str, "%.2f", CONC);
        strcat(filename,conc_str);
        strcat(filename,"/");
        if (folder_created == false){
          check = mkdir(filename,0777);
          // check if directory is created or not
          if (!check){
            printf("Directory created\n");
            }
          else {
            printf("Unable to create directory, or the folder is already created, relax mate...\n");  
        }
        folder_created = true;
        }
        
        strcat(filename,sample_str);
        strcat(filename,"_timeseries.csv");

        writer = fopen(filename,"w");
        fprintf(writer, "Time,Vm,dVm/dt,Cai,INa,INaL,ICaL,IKs,IKr,IK1,Ito\n"); 
        for (int datapoint = 1; datapoint<datapoint_size; datapoint++){
        if (h_time[ sample_id + (datapoint * sample_size)] == 0.0) {break;}
          fprintf(writer,"%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf\n", // change this into string, or limit the decimal accuracy, so we can decrease filesize
          h_time[ sample_id + (datapoint * sample_size)],
          h_states[ sample_id + (datapoint * sample_size)],
          h_dt[ sample_id + (datapoint * sample_size)],
          h_cai_result[ sample_id + (datapoint * sample_size)],
          
          h_ina[ sample_id + (datapoint * sample_size)], 
          h_inal[ sample_id + (datapoint * sample_size)], 

          h_ical[ sample_id + (datapoint * sample_size)],
          h_iks[ sample_id + (datapoint * sample_size)], 

          h_ikr[ sample_id + (datapoint * sample_size)],
          h_ik1[ sample_id + (datapoint * sample_size)],

          h_ito[ sample_id + (datapoint * sample_size)]  
          );
        }
        fclose(writer);
      }

      printf("writing each biomarkers value... \n");
      // sample loop
        char conc_str[ENOUGH];
        char filename[500] = "./result/";
        // sprintf(sample_str, "%d", sample_id);
        sprintf(conc_str, "%.2f", CONC);
        strcat(filename,conc_str);
        strcat(filename,"/");
        // printf("creating %s... \n", filename);
        if (folder_created == false){
          check = mkdir(filename,0777);
          // check if directory is created or not
          if (!check){
            printf("Directory created\n");
            }
          else {
            printf("Unable to create directory, or the folder is already created, relax mate...\n");  
        }
        folder_created = true;
        }
        
        // strcat(filename,sample_str);
      strcat(filename,"_biomarkers.csv");

      writer = fopen(filename,"a");

      fprintf(writer, "sample,qnet,qInward,inal_auc,ical_auc,apd90,apd50,apd_tri,cad90,cad50,cad_tri,dvmdt_repol,vm_peak,vm_valley,vm_dia,ca_peak,ca_valley,ca_dia\n"); 
      for (int sample_id = 0; sample_id<sample_size; sample_id++){
        // printf("writing sample %d... \n",sample_id);
        
        fprintf(writer,"%d,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf\n", // change this into string, or limit the decimal accuracy, so we can decrease filesize
          sample_id,
          h_cipa_result[sample_id].qnet,
          0.5*((h_cipa_result[sample_id].ical_auc / ical_auc_control)+(h_cipa_result[sample_id].inal_auc / inal_auc_control)),
          h_cipa_result[sample_id].inal_auc,
          h_cipa_result[sample_id].ical_auc,
          
          h_cipa_result[sample_id].apd90,
          h_cipa_result[sample_id].apd50,
          h_cipa_result[sample_id].apd90 - h_cipa_result[sample_id].apd50,

          h_cipa_result[sample_id].cad90,
          h_cipa_result[sample_id].cad50,
          h_cipa_result[sample_id].cad90 - h_cipa_result[sample_id].cad50,

          h_cipa_result[sample_id].dvmdt_repol,
          h_cipa_result[sample_id].vm_peak,
          h_cipa_result[sample_id].vm_valley,
          h_cipa_result[sample_id].vm_dia,

          h_cipa_result[sample_id].ca_peak,
          h_cipa_result[sample_id].ca_valley,
          h_cipa_result[sample_id].ca_dia

      //      temp_result[sample_id].qnet = 0.;
      // temp_result[sample_id].inal_auc = 0.;
      // temp_result[sample_id].ical_auc = 0.;

      // temp_result[sample_id].dvmdt_repol = -999;
      // temp_result[sample_id].dvmdt_max = -999;
      // temp_result[sample_id].vm_peak = -999;
      // temp_result[sample_id].vm_valley = d_STATES[(sample_id * num_of_states) +V];
      // temp_result[sample_id].vm_dia = -999;

      // temp_result[sample_id].apd90 = 0.;
      // temp_result[sample_id].apd50 = 0.;
      // temp_result[sample_id].ca_peak = -999;
      // temp_result[sample_id].ca_valley = d_STATES[(sample_id * num_of_states) +cai];
      // temp_result[sample_id].ca_dia = -999;
      // temp_result[sample_id].cad90 = 0.;
      // temp_result[sample_id].cad50 = 0.;
          );

      }
      fclose(writer);

      toc();
      
      return 0;
    }








    ////////// find cache mode (in silico code) //////////
  else{
    printf("In-silico mode, creating cache file because we don't have that yet, or is_time_series is intentionally false \n\n");
    double *d_ic50;
    double *d_cvar;
    double *d_ALGEBRAIC;
    double *d_CONSTANTS;
    double *d_RATES;
    double *d_STATES;

    // not used, only to satisfy the parameters of the GPU regulator's function
    double *d_STATES_cache;
    double *time;
    double *dt;
    double *states;
    double *cai_result;
    double *ical;
    double *inal;
    double *ina;
    double *ito;
    double *ikr;
    double *iks;
    double *ik1;

    double *d_STATES_RESULT;
    double *d_all_states;

    cipa_t *temp_result, *cipa_result;

    // snprintf(buffer, sizeof(buffer),
    //   "./drugs/bepridil/IC50_samples.csv"
    //   // "./drugs/bepridil/IC50_optimal.csv"
    //   // "./IC50_samples.csv"
    //   );
    int sample_size = get_IC50_data_from_file(p_param->hill_file, ic50);
    if(sample_size == 0)
        printf("Something problem with the IC50 file!\n");
    // else if(sample_size > 2000)
    //     printf("Too much input! Maximum sample data is 2000!\n");
    printf("Sample size: %d\n",sample_size);
    hipSetDevice(p_param->gpu_index);
    printf("preparing GPU memory space \n");

    if(p_param->is_cvar == true){
      int cvar_sample = get_cvar_data_from_file(p_param->cvar_file,sample_size,cvar);
      printf("Reading: %d Conductance Variability samples\n",cvar_sample);
    }

    hipMalloc(&d_ALGEBRAIC, num_of_algebraic * sample_size * sizeof(double));
    hipMalloc(&d_CONSTANTS, num_of_constants * sample_size * sizeof(double));
    hipMalloc(&d_RATES, num_of_rates * sample_size * sizeof(double));
    hipMalloc(&d_STATES, num_of_states * sample_size * sizeof(double));

    hipMalloc(&d_p_param,  sizeof(param_t));

    // prep for 1 cycle plus a bit (7000 * sample_size)
    hipMalloc(&temp_result, sample_size * sizeof(cipa_t));
    hipMalloc(&cipa_result, sample_size * sizeof(cipa_t));

    hipMalloc(&d_STATES_RESULT, (num_of_states+1) * sample_size * sizeof(double)); // for cache file
    hipMalloc(&d_all_states, num_of_states * sample_size * p_param->find_steepest_start * sizeof(double)); // for each sample 

    printf("Copying sample files to GPU memory space \n");
    hipMalloc(&d_ic50, sample_size * 14 * sizeof(double));
    // if(p_param->is_cvar == true) hipMalloc(&d_cvar, sample_size * 18 * sizeof(double));
    // hipMalloc(&d_cvar, sample_size * 18 * sizeof(double));
    hipMalloc(&d_cvar, sizeof(double));
    
    hipMemcpy(d_ic50, ic50, sample_size * 14 * sizeof(double), hipMemcpyHostToDevice);
    // if(p_param->is_cvar == true) hipMemcpy(d_cvar, cvar, sample_size * 18 * sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(d_cvar, cvar, sample_size * 18 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_p_param, p_param, sizeof(param_t), hipMemcpyHostToDevice);

    // // Get the maximum number of active blocks per multiprocessor
    // hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, do_drug_sim_analytical, threadsPerBlock);

    // // Calculate the total number of blocks
    // int numTotalBlocks = numBlocks * cudaDeviceGetMultiprocessorCount();

    tic();
    printf("Timer started, doing simulation.... \n GPU Usage at this moment: \n");
    int thread;
    if (sample_size>=32){
      thread = 32;
    }
    else thread = sample_size;
    // int block = int(ceil(sample_size*1.0/thread)+1);
    int block = (sample_size + thread - 1) / thread;
    // int block = (sample_size + thread - 1) / thread;
    if(gpu_check(15 * sample_size * sizeof(double) + sizeof(param_t)) == 1){
      printf("GPU memory insufficient!\n");
      return 0;
    }
    printf("Sample size: %d\n",sample_size);
    hipSetDevice(p_param->gpu_index);
    printf("\n   Configuration: \n\n\tblock\t||\tthread\n---------------------------------------\n  \t%d\t||\t%d\n\n\n", block,thread);
    // initscr();
    // printf("[____________________________________________________________________________________________________]  0.00 %% \n");

    kernel_DrugSimulation<<<block,thread>>>(d_ic50, d_cvar, d_CONSTANTS, d_STATES, d_STATES_cache, d_RATES, d_ALGEBRAIC, 
                                              d_STATES_RESULT, d_all_states,
                                              time, states, dt, cai_result,
                                              ina, inal, 
                                              ical, ito,
                                              ikr, iks, 
                                              ik1,
                                              sample_size,
                                              temp_result, cipa_result,
                                              d_p_param
                                              );
                                      //block per grid, threads per block
    // endwin();
    
    hipDeviceSynchronize();
    

    printf("allocating memory for computation result in the CPU, malloc style \n");
    double *h_states, *h_all_states;
    cipa_t *h_cipa_result;

    h_states = (double *)malloc((num_of_states+1) * sample_size * sizeof(double)); //cache file
    h_all_states = (double *)malloc( (num_of_states) * sample_size * p_param->find_steepest_start * sizeof(double)); //all core
    h_cipa_result = (cipa_t *)malloc(sample_size * sizeof(cipa_t));
    printf("...allocating for all states, all set!\n");

    ////// copy the data back to CPU, and write them into file ////////
    printf("copying the data back to the CPU \n");

    hipMemcpy(h_cipa_result, cipa_result, sample_size * sizeof(cipa_t), hipMemcpyDeviceToHost);
    hipMemcpy(h_states, d_STATES_RESULT, sample_size * (num_of_states+1) *  sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_all_states, d_all_states, (num_of_states) * sample_size  * p_param->find_steepest_start  *  sizeof(double), hipMemcpyDeviceToHost);

    FILE *writer;
    int check;
    bool folder_created = false;

    
    // char sample_str[ENOUGH];
    char conc_str[ENOUGH];
    char filename[500] = "./result/";
    sprintf(conc_str, "%.2f", CONC);
    strcat(filename,conc_str);
    // strcat(filename,"_steepest");
      if (folder_created == false){
        check = mkdir(filename,0777);
        // check if directory is created or not
        if (!check){
          printf("Directory created\n");
          }
        else {
          printf("Unable to create directory, or the folder is already created, relax mate...\n");  
      }
      folder_created = true;
      }
      
    // strcat(filename,sample_str);
    strcat(filename,".csv");
    printf("writing to %s ... \n", filename);
    writer = fopen(filename,"w");
    // sample loop
    for (int sample_id = 0; sample_id<sample_size; sample_id++){
      // writer = fopen(filename,"a"); // because we have multiple fwrites
      fprintf(writer,"%d,",sample_id); // write core number at the front
      for (int datapoint = 0; datapoint<num_of_states; datapoint++){
       // if (h_time[ sample_id + (datapoint * sample_size)] == 0.0) {continue;}
        fprintf(writer,"%.5f,", // change this into string, or limit the decimal accuracy, so we can decrease filesize
        h_states[(sample_id * (num_of_states+1)) + datapoint]
        );
      }
        // fprintf(writer,"%lf,%lf\n", // write last data
        // h_states[(sample_id * num_of_states+1) + num_of_states],
        // h_states[(sample_id * num_of_states+1) + num_of_states+1]
        // );
        fprintf(writer,"%.5f\n", h_states[(sample_id * (num_of_states+1))+num_of_states] );
        // fprintf(writer, "\n");

      // fclose(writer);
    }
     fclose(writer);

    // // FILE *writer;
    // // int check;
    // // bool folder_created = false;

    // printf("writing each core value... \n");
    // // sample loop
    // for (int sample_id = 0; sample_id<sample_size; sample_id++){
    //   // printf("writing sample %d... \n",sample_id);
    //   char sample_str[ENOUGH];
    //   char conc_str[ENOUGH];
    //   char filename[500] = "./result/";
    //   sprintf(sample_str, "%d", sample_id);
    //   sprintf(conc_str, "%.2f", CONC);
    //   strcat(filename,conc_str);
    //   strcat(filename,"/");
    //   // printf("creating %s... \n", filename);
    //   if (folder_created == false){
    //     check = mkdir(filename,0777);
    //     // check if directory is created or not
    //     if (!check){
    //       printf("Directory created\n");
    //       }
    //     else {
    //       printf("Unable to create directory, or the folder is already created, relax mate...\n");  
    //   }
    //   folder_created = true;
    //   }
      
    //   strcat(filename,sample_str);
    //   strcat(filename,".csv");

    //   writer = fopen(filename,"w");
    //   for (int pacing = 0; pacing < p_param->find_steepest_start; pacing++){ //pace loop
    //    // if (h_time[ sample_id + (datapoint * sample_size)] == 0.0) {continue;}
    //     for(int datapoint = 0; datapoint < num_of_states; datapoint++){ // each data loop
    //     fprintf(writer,"%lf,",h_all_states[((sample_id * num_of_states)) + ((sample_size) * pacing) + datapoint]);
    //     // fprintf(writer,"%lf,",h_all_states[((sample_id * num_of_states))+ datapoint]);
    //     } 
    //     // fprintf(writer,"%d",p_param->find_steepest_start + pacing);
    //     fprintf(writer,"%d\n",pacing + (p_param->pace_max - p_param->find_steepest_start)+1 );

    //   }
    //   fclose(writer);
    // }

    toc();
    
    return 0;

    }
   	
}
